#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <opencv2/cudaimgproc.hpp>
#include <opencv2/cudawarping.hpp>
#include <hip/hip_runtime.h>
#include <SFML/Audio.hpp>
#include <thread>
#include <chrono>
#include <sstream>
#include <mutex>
#include <iostream>
#include <vector>
#include <string>
#include <unistd.h>
#include <ctime>

constexpr float volume = 70.0f;
constexpr float speed = 1.0f;
constexpr int fps_value = 1;
constexpr int HEIGHT = 240; // 画像の高さ
constexpr float sleep_value = 2; //待機時間
const std::string FILENAME = "idol.webm";

__global__ void quantize_kernel(const cv::cuda::PtrStepSz<uchar3> input_image, int* output_quantized) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < input_image.cols && y < input_image.rows) {
        uchar3 pixel = input_image(y, x);

        int quantized_red = (pixel.x / 3) * 3;
        int quantized_green = (pixel.y / 3) * 3;
        int quantized_blue = (pixel.z / 3) * 3;

        output_quantized[3 * (y * input_image.cols + x) + 0] = quantized_red;
        output_quantized[3 * (y * input_image.cols + x) + 1] = quantized_green;
        output_quantized[3 * (y * input_image.cols + x) + 2] = quantized_blue;
    }
}


cv::cuda::GpuMat resize_cuda(const cv::cuda::GpuMat& d_image) {
    int old_width = d_image.cols;
    int old_height = d_image.rows;
    float aspect_ratio = static_cast<float>(old_width) / static_cast<float>(old_height);
    int new_width = static_cast<int>(aspect_ratio * HEIGHT * 2.5);
    cv::cuda::GpuMat d_resized_image;
    cv::cuda::resize(d_image, d_resized_image, cv::Size(new_width, HEIGHT));
    return d_resized_image;
}

std::string modify_cuda(const cv::cuda::GpuMat& d_image) {
    int* d_quantized;
    hipMalloc(&d_quantized, d_image.rows * d_image.cols * 3 * sizeof(int));    // 結果を格納するGPUメモリを確保
    dim3 blockSize(16, 16);
    dim3 gridSize((d_image.cols + blockSize.x - 1) / blockSize.x, 
                  (d_image.rows + blockSize.y - 1) / blockSize.y);
    quantize_kernel<<<gridSize, blockSize>>>(d_image, d_quantized);
    int* h_quantized = new int[d_image.rows * d_image.cols * 3];
    hipMemcpy(h_quantized, d_quantized, d_image.rows * d_image.cols * 3 * sizeof(int), hipMemcpyDeviceToHost);

    std::ostringstream oss;
    oss << "\033[H";
    int prev_red = -1, prev_green = -1, prev_blue = -1;
    for (int i = 0; i < d_image.rows; ++i) {
        for (int j = 0; j < d_image.cols; ++j) {
            int idx = 3 * (i * d_image.cols + j);
            int quantized_red = h_quantized[idx + 0];
            int quantized_green = h_quantized[idx + 1];
            int quantized_blue = h_quantized[idx + 2];
            if (quantized_red != prev_red || quantized_green != prev_green || quantized_blue != prev_blue) {
                oss << "\033[48;2;" << quantized_red << ";" << quantized_green << ";" << quantized_blue << "m";
                prev_red = quantized_red;
                prev_green = quantized_green;
                prev_blue = quantized_blue;
            }
            oss << " ";
        }
        oss << "\n";
    }
    oss << "\033[0m";
    hipFree(d_quantized);
    delete[] h_quantized;

    return oss.str();
}

std::string doProcess(const cv::Mat& image) {
    cv::cuda::GpuMat d_image;
    d_image.upload(image);
    cv::cuda::GpuMat d_resized_image;
    d_resized_image = resize_cuda(d_image);
    std::string result = modify_cuda(d_resized_image);
    return result;
}

int main() {
    std::string commands = "ffmpeg -y -hwaccel cuda -i " + FILENAME + " -vn output.wav";
    std::thread th([&commands] {
        system(commands.c_str());
    });
    cv::VideoCapture vidObj(FILENAME);
    std::ios_base::sync_with_stdio(false);
    std::cin.tie(nullptr);
    if (!vidObj.isOpened()) {
        std::cerr << "Error: Video file not opened" << std::endl;
        return 1;
    }
    std::vector<std::string> frames;
    std::mutex frames_mutex;
    cv::Mat image;
    FILE *fp;
    int frame_count = static_cast<int>(vidObj.get(cv::CAP_PROP_FRAME_COUNT));
    fp = fopen("output.txt", "w");
    std::thread cv_thred([&frame_count, &frames, &vidObj, &image, &frames_mutex, &fp](){
        for (size_t i = 0; i < frame_count; i += fps_value){
            fprintf(fp, "frame_count = %ld\n", i);
            if(!vidObj.read(image))break;
            std::string frame = doProcess(image);
            if (!frame.empty()) {
                std::lock_guard<std::mutex> lock(frames_mutex);
                frames.emplace_back(frame);
            }
            for (int j = 1; j < fps_value; j++) {
                if (!vidObj.grab()) break;
            }
            fprintf(fp, "frames.size() = %ld\n",frames.size());
        }
        vidObj.release();
        fprintf(fp, "end_cv2\n");
    });
    th.join();
    while (frames.size() < (frame_count / fps_value) / sleep_value) {
        std::this_thread::sleep_for(std::chrono::milliseconds(100));
    }
    system("clear");
    float fps = vidObj.get(cv::CAP_PROP_FPS) / fps_value * speed;
    sf::Music music;
    if (!music.openFromFile("output.wav")) {
        std::cerr << "Error loading audio file" << std::endl;
        return -1;
    }
    music.setPitch(speed);
    music.setVolume(volume);
    music.play();
    auto start_time = std::chrono::high_resolution_clock::now();
    std::thread display_thread([&frames, &start_time, &frames_mutex, fps, frame_count, &fp]() {
        for (size_t i = 0; i < ((frame_count / fps_value) -2); ++i) {
            auto current_time = std::chrono::high_resolution_clock::now();
            std::chrono::duration<double> elapsed_time = current_time - start_time;
            int expected_frame_index = static_cast<int>(elapsed_time.count() * fps);
            std::lock_guard<std::mutex> lock(frames_mutex);
            while (i < expected_frame_index && i < (frame_count / fps_value) && i < frames.size()) {
                ++i;
            }
            auto frame_start_time = std::chrono::high_resolution_clock::now();
            {
                if (i < frames.size() && !frames[i].empty()) {
                    write(STDOUT_FILENO, frames[i].c_str(), frames[i].size());
                    frames[i].clear();
                    frames[i].shrink_to_fit();
                } else {
                    fprintf(fp, "frame = %ld, frames.size() = %ld\n", i, frames.size());
                }
            }
            auto frame_end_time = std::chrono::high_resolution_clock::now();
            std::chrono::duration<double> processing_time = frame_end_time - frame_start_time;
            double sleep_time = (1.0 / fps) - processing_time.count();
            if (sleep_time > 0) {
                std::this_thread::sleep_for(std::chrono::duration<double>(sleep_time));
            }
            fprintf(fp, "display_frame = %ld, processing_time = %f, sleep_time = %f, frames.size - i = %ld\n", i, processing_time.count(), sleep_time, frames.size() - i);
        }
    });

    display_thread.join();
    cv_thred.join();
    music.stop();
    system("clear");
    printf("end_display\n");
    fprintf(fp, "end\n");
    fclose(fp);
    return 0;
}