#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <opencv2/cudaimgproc.hpp>
#include <opencv2/cudawarping.hpp>
#include <hip/hip_runtime.h>

constexpr float volume = 70.0f;
constexpr float speed = 1.0f;
constexpr int fps_value = 1;
constexpr int HEIGHT = 240; // 画像の高さ
constexpr float sleep_value = 4; //待機時間

__global__ void quantize_kernel(const cv::cuda::PtrStepSz<uchar3> input_image, int* output_quantized) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < input_image.cols && y < input_image.rows) {
        uchar3 pixel = input_image(y, x);
        int quantized_red = (pixel.x / 3) * 3;
        int quantized_green = (pixel.y / 3) * 3;
        int quantized_blue = (pixel.z / 3) * 3;

        int idx = 3 * (y * input_image.cols + x);
        output_quantized[idx + 0] = quantized_red;
        output_quantized[idx + 1] = quantized_green;
        output_quantized[idx + 2] = quantized_blue;
    }
}

cv::cuda::GpuMat resize_cuda(const cv::cuda::GpuMat& d_image) {
    int old_width = d_image.cols;
    int old_height = d_image.rows;
    float aspect_ratio = static_cast<float>(old_width) / static_cast<float>(old_height);
    int new_width = static_cast<int>(aspect_ratio * HEIGHT * 2.5);

    cv::cuda::GpuMat d_resized_image;
    cv::cuda::resize(d_image, d_resized_image, cv::Size(new_width, new_height));

    return d_resized_image;
}

std::string modify_cuda(const cv::cuda::GpuMat& d_image) {
    int* d_quantized;
    hipMalloc(&d_quantized, d_image.rows * d_image.cols * 3 * sizeof(int));    // 結果を格納するGPUメモリを確保
    dim3 blockSize(16, 16);
    dim3 gridSize((d_image.cols + blockSize.x - 1) / blockSize.x, 
                  (d_image.rows + blockSize.y - 1) / blockSize.y);
    quantize_kernel<<<gridSize, blockSize>>>(d_image, d_quantized);
    int* h_quantized = new int[d_image.rows * d_image.cols * 3];
    hipMemcpy(h_quantized, d_quantized, d_image.rows * d_image.cols * 3 * sizeof(int), hipMemcpyDeviceToHost);

    std::ostringstream oss;
    oss << "\033[H";
    int prev_red = -1, prev_green = -1, prev_blue = -1;
    for (int i = 0; i < d_image.rows; ++i) {
        for (int j = 0; j < d_image.cols; ++j) {
            int idx = 3 * (i * d_image.cols + j);
            int quantized_red = h_quantized[idx + 0];
            int quantized_green = h_quantized[idx + 1];
            int quantized_blue = h_quantized[idx + 2];

            if (quantized_red != prev_red || quantized_green != prev_green || quantized_blue != prev_blue) {
                oss << "\033[48;2;" << quantized_red << ";" << quantized_green << ";" << quantized_blue << "m";
                prev_red = quantized_red;
                prev_green = quantized_green;
                prev_blue = quantized_blue;
            }
            oss << " ";
        }
        oss << "\n";
    }
    oss << "\033[0m";
    hipFree(d_quantized);
    delete[] h_quantized;

    return oss.str();
}

int main() {
    cv::Mat image = cv::imread("test_image.jpg");
    if (image.empty()) {
        std::cerr << "画像の読み込みに失敗しました。" << std::endl;
        return -1;
    }
    cv::cuda::GpuMat d_image;
    d_image.upload(image);

    cv::cuda::GpuMat d_resized_image;
    d_resized_image = resize_cuda(d_image);
    std::string result = modify_cuda(d_resized_image);
    std::cout << result << std::endl;

    return 0;
}

